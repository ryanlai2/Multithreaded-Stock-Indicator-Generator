#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <io.h>
#include <string.h>

typedef struct daily_data
{
	float open;
	float high;
	float low;
	float close;
	float volume;
	float adj_close;
	char date[12];
}data_t; // every line has 69 chars (68 + 1 * '\n')

__global__ void parseKernel(char * data, data_t * trading_data)
{
	int i = threadIdx.x; 
	char temp[69]; 
	memcpy(temp, data + i * 69, 69); 
	temp[10] = 0;
	temp[20] = 0;
	temp[30] = 0;
	temp[40] = 0;
	temp[50] = 0;
	temp[58] = 0;
	temp[68] = 0;
	memcpy(trading_data[i].data, temp, 11); 
	trading_data[i].open = strtof(temp+11);
	trading_data[i].high = strtof(temp + 21);
	trading_data[i].low = strtof(temp + 31);
	trading_data[i].close = strtof(temp + 41);
	trading_data[i].volume = atoi(temp + 51);
	trading_data[i].adj_close = strtof(temp + 58);
}

__global__ void computation(data_t * trading_data)
{

}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
	memcpy(c + i, a + i, 4);
    //c[i] = a[i] + b[i];
}



__shared__ char data[16384];

int main()
{
	FILE * f = fopen("table.csv", "r");
	int d; 
	size_t f_size; 
	f_size = fread(data, 1, 16384,  f);
	printf("length of string is %ld", strlen(data));
	unsigned int days = (int)strlen(data) / 69;
	data_t * trading_data = (data_t *) malloc(days * sizeof(data_t));

	for (int i = 0; i < 10300; i++)
	{
		//putc(data[i], stdout);
		if (data[i] == ',')
		{
			printf("%i is a comma\n", i); 
		}
		if (data[i] == '\n') break; 
	}

	scanf("%d", &d); 
	return 0;
	/*
	const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
	//int varvar; 
	//scanf("%d", &varvar);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;*/
}

// Helper function for using CUDA to add vectors in parallel.

hipError_t parse_cuda(char * data, data_t * trading_data, unsigned int days)
{
	char *dev_data = NULL;
	data_t * dev_trading_data = NULL; 

	hipMalloc((void**) &dev_data, strlen(data));
	hipMalloc((void**) & dev_trading_data, days * sizeof(trading_data)); 
	hipError_t cuaStatus; 
	hipMemcpy(dev_data, data, strlen(data), hipMemcpyHostToDevice);  
	parseKernel <<<1, days >>>(dev_data, dev_trading_data);
	hipDeviceSynchronize(); 
	hipMemcpy(trading_data, dev_trading_data, sizeof(data_t)*days , hipMemcpyDeviceToHost); 
	hipFree(dev_data);
	hipFree(dev_trading_data);
	return hipSuccess;
}

/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/