#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <io.h>
#include <string.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>


#include "C:\Program Files\boost_1_62_0\boost\asio\buffer.hpp"
#include "C:\Program Files\boost_1_62_0\boost\asio.hpp"
#include "C:\Program Files\boost_1_62_0\boost\system\error_code.hpp"
#include "Dependencies\glew\glew.h"
#include "Dependencies\freeglut\freeglut.h"



using namespace boost::asio::ip;
std::string stock;
std::fstream temp_stream;

const int window_height = 720; 
const int window_width = 1280; 
int days_read = 0; 
float max_close_price = 0.0; 
float min_close_price = 9999999999.99; 
float max_result = -999999999.99, min_result = 999999999.99; 



class data_t
{
public:
	float open;
	float high;
	float low;
	float close;
	float volume;
	float adj_close;
	char date[12];
};


std::vector<data_t> prices;

float * results;

GLuint program;
GLint attribute_coord2d;

void print_text()
{
	glColor3f(0, 0, 0);
	glRasterPos2f(-0.5, -0.5);
	int i;
	char price_buffer[10]; 
	char idx_buffer[64]; 
	char label[64]; 
	sprintf(label, "PRICE OF %s", stock.c_str());

	glRasterPos2f(-0.145,0.9);
	for (size_t j = 0; j < strlen(label); j++)
	{
		glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, label[j]);
	}
	
	glColor3f(1.0, 0, 0);
	glRasterPos2f(-0.16, -0.1);
	char label2[64];
	sprintf(label2, "ON BALANCE VOLUME OF %s", stock.c_str());
	for (size_t j = 0; j < strlen(label2); j++)
	{
		glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, label2[j]);
	}
	
	for (float i = 0; i < 7; i++)
	{
		float p = min_close_price + (max_close_price - min_close_price)*i / 6.0; 
		sprintf(price_buffer, "%.2f", p);
		glColor3f(0, 0, 0);

		glRasterPos2f(-0.9, 0.2 + 0.1 * i);
		for (size_t j = 0; j < strlen(price_buffer); j++)
		{
			glutBitmapCharacter(GLUT_BITMAP_HELVETICA_10, price_buffer[j]);
		}

		p = min_result + (max_result - min_result)*i / 6.0;
		sprintf(idx_buffer, "%.2f", p);
		glColor3f(1.0, 0, 0);
		glRasterPos2f(-0.98, -0.8 + 0.1 * i);
		for (size_t j = 0; j < strlen(idx_buffer); j++)
		{
			glutBitmapCharacter(GLUT_BITMAP_HELVETICA_10, idx_buffer[j]);
		}

	}
	
	int day, idx = 0; 
	for (float f = -0.8; f <= 1.0; f += 0.4)
	{
		day = (days_read-1) / 4 * idx++; 
		glColor3f(0, 0, 0);
		glRasterPos2f(f-0.05, 0.12);
		for (size_t j = 0; j < strlen(prices[day].date); j++)
		{
			glutBitmapCharacter(GLUT_BITMAP_HELVETICA_10, prices[day].date[j]);
		}

		glColor3f(1.0, 0, 0);
		glRasterPos2f(f - 0.05, -0.88);
		for (size_t j = 0; j < strlen(prices[day].date); j++)
		{
			glutBitmapCharacter(GLUT_BITMAP_HELVETICA_10, prices[day].date[j]);
		}
	}
	
}
struct Point
{
	float x, y;
	unsigned char r, g, b, a;
};
std::vector< Point > points;
std::vector< Point > upper_coor;
std::vector <Point> result_points; 
std::vector <Point> lower_coor; 
void display(void)
{
	glClearColor(1.0, 1.0, 1.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	//glOrtho(-50, 50, -50, 50, -1, 1);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	// draw
	glColor3ub(255, 255, 255);
	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glVertexPointer(2, GL_FLOAT, sizeof(Point), &points[0].x);
	glColorPointer(4, GL_UNSIGNED_BYTE, sizeof(Point), &points[0].r);
	glPointSize(3.0);
	glDrawArrays(GL_LINE_STRIP, 0, points.size());
	glDisableClientState(GL_VERTEX_ARRAY);
	glDisableClientState(GL_COLOR_ARRAY);


	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glVertexPointer(2, GL_FLOAT, sizeof(Point), &upper_coor[0].x);
	glColorPointer(4, GL_UNSIGNED_BYTE, sizeof(Point), &upper_coor[0].r);
	glPointSize(3.0);
	glDrawArrays(GL_LINE_STRIP, 0, upper_coor.size());
	glDisableClientState(GL_VERTEX_ARRAY);
	glDisableClientState(GL_COLOR_ARRAY);



	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glVertexPointer(2, GL_FLOAT, sizeof(Point), &result_points[0].x);
	glColorPointer(4, GL_UNSIGNED_BYTE, sizeof(Point), &result_points[0].r);
	glPointSize(3.0);
	glDrawArrays(GL_LINE_STRIP, 0, result_points.size());
	glDisableClientState(GL_VERTEX_ARRAY);
	glDisableClientState(GL_COLOR_ARRAY);



	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glVertexPointer(2, GL_FLOAT, sizeof(Point), &lower_coor[0].x);
	glColorPointer(4, GL_UNSIGNED_BYTE, sizeof(Point), &lower_coor[0].r);
	glPointSize(3.0);
	glDrawArrays(GL_LINE_STRIP, 0, lower_coor.size());
	glDisableClientState(GL_VERTEX_ARRAY);
	glDisableClientState(GL_COLOR_ARRAY);



	print_text();

	glFlush();
	glutSwapBuffers();
}

void reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}





__global__ void GPU_computation(data_t * prices, float * results, int offset)
{
	int i = threadIdx.x + offset;
	if (i == 0) results[i] = 0; 
	else
	{
		if (prices[i].close > prices[i - 1].close) results[i] = prices[i].volume;
		else if (prices[i].close < prices[i - 1].close) results[i] = -prices[i].volume;
		else results[i] = 0; 
	}
}

hipError_t generate_cuda(std::vector<data_t> &prices, float * results);


void read_from_internet()
{
	std::cout << "Enter the symbol of a stock in capital letters" << std::endl;
	std::cout << "Enter the starting and ending data in the following format:" << std::endl;
	std::cout << "fasdf " << std::endl;
	std::cout << "Invalid input has undefined behavior" << std::endl;
	std::cin >> stock;
	int start_month, start_date, start_year, end_month, end_date, end_year;
	scanf("%d %d %d %d %d %d", &start_month, &start_date, &start_year, &end_month, &end_date, &end_year);
	std::cout << start_month << " " << start_date <<" " << start_year <<" " << end_month << " " << end_date<<" "  << end_year << std::endl; 
	stock[stock.length()] = 0; 

	start_month--; end_month--; 
	//int start_month = 1, start_date = 13, start_year = 2016, end_month = 10, end_date = 13, end_year = 2016;

	boost::system::error_code error;

	boost::asio::io_service io_service;
	tcp::socket socket(io_service);
	tcp::resolver resolver(io_service);
	tcp::resolver::query query("chart.finance.yahoo.com", "http");
	tcp::resolver::iterator i = resolver.resolve(query);
	boost::asio::connect(socket, i);
	boost::asio::streambuf request, response;
	std::ostream request_stream(&request);
	request_stream << "GET /table.csv?s=" << stock << "&a=" << start_month << "&b="
		<< start_date << "&c=" << start_year << "&d=" << end_month << "&e="
		<< end_date << "&f=" << "2016" << "&g=d&ignore=.csv HTTP/1.1\r\nHost: chart.finance.yahoo.com\r\n\r\n";

	boost::asio::write(socket, request);
	std::istream response_stream(&response);


	temp_stream.open("temp.txt", std::fstream::in | std::fstream::out | std::fstream::trunc);

	char buffer[4096];
	char start_sign[8];
	int start_flag = 0;
	sprintf(start_sign, "%d-", end_year);
	char temp[10];
	temp[8] = 0;
	for (int k = 0; k < 1000000; k++)
	{
		size_t x = boost::asio::read_until(socket, response, "\n", error);
		response_stream.getline(buffer, x);
		std::cout << buffer << std::endl;
		if (strlen(buffer) < 5 && start_flag) break;
		if (!start_flag) {
			memcpy(temp, buffer, 5);
			temp[5] = 0;
			if (strcmp(temp, start_sign) == 0) {
				start_flag = 1;
				temp_stream << buffer << "\n";
				days_read++; 
				std::cout << buffer << std::endl;
			}
		}
		//else std::cout << buffer << std::endl;
		else { temp_stream << buffer << "\n"; days_read++;  std::cout << buffer << std::endl;
		}

		//memcpy(temp, buffer + 2, 8);

	}
	temp_stream.seekp(0);
}

void read_prices(std::vector<data_t> & prices)
{
	data_t * temp = new data_t();
	std::string str;
	const char * ptr = NULL;
	int i = 0;
	while (std::getline(temp_stream, str))
	{
		ptr = str.c_str();

		memcpy(temp->date, ptr, 10);

		temp->date[10] = 0;
		ptr = strchr(ptr, ',') + 1;
		temp->open = atof(ptr);

		ptr = strchr(ptr, ',') + 1;
		temp->high = atof(ptr);

		ptr = strchr(ptr, ',') + 1;
		temp->low = atof(ptr);

		ptr = strchr(ptr, ',') + 1;
		temp->close = atof(ptr);

		if (max_close_price < temp->close) max_close_price = temp->close; 
		if (min_close_price > temp->close) min_close_price = temp->close; 

		ptr = strchr(ptr, ',') + 1;
		temp->volume = atoi(ptr);

		ptr = strchr(ptr, ',') + 1;
		temp->adj_close = atof(ptr);


		prices.push_back(*temp);
	}
	free(temp);
	temp_stream.close();
	//fb.close();
	temp = NULL;
	return;
}

int glut_window(int argc, char ** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_DOUBLE);

	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Random Points");

	glutDisplayFunc(display);
	glutReshapeFunc(reshape);

	// populate points
	for (size_t i = 0; i < days_read; ++i)
	{
		Point pt;
		pt.x = (float) i / (float) days_read * 1.6 - 0.8;
		pt.y = (prices[i].close - min_close_price) / (max_close_price - min_close_price) * 0.6 + 0.2;
		pt.r = 0;
		pt.g = 0;
		pt.b = 0;
		pt.a = 255;
		points.push_back(pt);
	}
	for (int l = 0; l < 1; l++) {
		Point pt1, pt2, pt3;

		pt1.x = -0.85;
		pt1.y = 0.85;
		pt1.r = 0;
		pt1.g = 0;
		pt1.b = 0;
		pt1.a = 255;

		pt2.x = -0.85;
		pt2.y = 0.15;
		pt2.r = 0;
		pt2.g = 0;
		pt2.b = 0;
		pt2.a = 255;

		pt3.x = 0.85;
		pt3.y = 0.15;
		pt3.r = 0;
		pt3.g = 0;
		pt3.b = 0;
		pt3.a = 255;
		upper_coor.push_back(pt1);
		upper_coor.push_back(pt2);
		upper_coor.push_back(pt3);
	}
	for (size_t i = 0; i < days_read; ++i)
	{
		Point pt;
		pt.x = (float)i / (float)days_read * 1.6 - 0.8;
		pt.y = (results[i] - min_result) / (max_result - min_result) * 0.6 - 0.8;
		pt.r = 255;
		pt.g = 0;
		pt.b = 0;
		pt.a = 255;
		result_points.push_back(pt);
	}

	for (int l = 0; l < 1; l++) {
		Point pt1, pt2, pt3;

		pt1.x = -0.85;
		pt1.y = -0.15;
		pt1.r = 255;
		pt1.g = 0;
		pt1.b = 0;
		pt1.a = 255;

		pt2.x = -0.85;
		pt2.y = -0.85;
		pt2.r = 255;
		pt2.g = 0;
		pt2.b = 0;
		pt2.a = 255;

		pt3.x = 0.85;
		pt3.y = -0.85;
		pt3.r = 255;
		pt3.g = 0;
		pt3.b = 0;
		pt3.a = 255;
		lower_coor.push_back(pt1);
		lower_coor.push_back(pt2);
		lower_coor.push_back(pt3);
	}




	glutMainLoop();
	return 0;
}

void process_results()
{
	// for OBV
	for (int i = 1; i < prices.size(); i++) {
		results[i] += results[i - 1]; 
		if (results[i] > max_result) max_result = results[i]; 
		if (results[i] < min_result) min_result = results[i]; 
	}
}

int main(int argc, char** argv)
{

	read_from_internet();
	
	read_prices(prices);
	std::reverse(prices.begin(), prices.end());
	results = (float*) calloc(prices.size(), sizeof(float));

	generate_cuda(prices, results);



	std::cout << "the size of the vector is" << prices.size() << std::endl;
	process_results(); 

	std::cout << results[5] << " " << results[120] << " " << std::endl;
	int d; 
	printf("finished\n");

	glut_window(argc, argv);

	scanf("%d", &d); 
	 


	return 0;
}


// Helper function for using CUDA to add vectors in parallel.

hipError_t generate_cuda(std::vector<data_t> &prices, float * results)
{
	data_t * price_array = NULL;
	size_t days = prices.size();
	float * dev_results = NULL; 
	hipMalloc((void**)&price_array, days * sizeof(data_t));
	hipMalloc((void**)&dev_results, days*sizeof(float));
	hipMemcpy(price_array, &(prices[0]), days * sizeof(data_t), hipMemcpyHostToDevice);
	hipError_t cudaStatus; 
	for (size_t i = 0; i < days; i += 250)
	{
		if (days - i >= 250)
		{
			GPU_computation << <1, 250 >> >(price_array, dev_results, (int)i);
			hipDeviceSynchronize();
		}
		else
		{
			GPU_computation << <1, days-i >> >(price_array, dev_results, (int)i);
			hipDeviceSynchronize();
		}
	}
	hipMemcpy(results, dev_results, sizeof(float)*days , hipMemcpyDeviceToHost); 
	hipFree(dev_results);
	hipFree(price_array);
	return hipSuccess;
}

